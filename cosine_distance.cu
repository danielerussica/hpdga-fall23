#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>
#include <assert.h>
#include <inttypes.h>

// TODO:
//      2. Implement add delta to min and check how many candidates are in the range
//      3. Implement candidate sorting (mabye do it on cpu? list should be small)

//      4. Explore other k selection methods



// -- OLD VERSION --
/*
    * Compute the cosine distance between two vectors
    * inspired from Cuda webinar on reduction kernel03 (mabye extend optimization to kernel04)
    * In this version, each block computes a single cosine distance between a variable ref point vs a query point, each block has "dim" threads
*/
__global__ void cdist(const float   * ref,
                        int           ref_nb,
                        const float * query,
                        int           query_nb,
                        int           dim,
                        int           query_index,
                        float       * d_gpu_dist){

    // we need 3 * blockDim * sizeof(float) shared memory
    extern __shared__ float smem[];

    // unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int tid = threadIdx.x;

    // dot product
    smem[tid]                = ((ref[(tid*ref_nb)+blockIdx.x]) * (query[(tid*query_nb)+query_index])) ;
    // denom_a
    smem[tid+blockDim.x]     = (ref[(tid*ref_nb)+blockIdx.x]) * (ref[(tid*ref_nb)+blockIdx.x]) ;
    // denom_b
    smem[tid+(2*blockDim.x)] = (query[(tid*query_nb)+query_index]) * (query[(tid*query_nb)+query_index]) ;

    if(smem[tid] == 0){
        printf("smem[%d]: %f\n", tid, smem[tid]);
        printf("smem[%d]: %f\n", tid+blockDim.x, smem[tid+blockDim.x]);
        printf("smem[%d]: %f\n", tid+(2*blockDim.x), smem[tid+(2*blockDim.x)]);
    }


    __syncthreads();

    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid < s){
            smem[tid] += smem[tid + s];
            smem[tid+blockDim.x] += smem[tid + s + blockDim.x];
            smem[tid+(2*blockDim.x)] += smem[tid + s + (2*blockDim.x)];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0){
        d_gpu_dist[(query_nb*blockIdx.x)+query_index] = smem[0] / (sqrt(smem[blockDim.x]) * sqrt(smem[2*blockDim.x]));
    }
}


// -- OLD VERSION --
/*
    * Compute the cosine distance between two vectors
    * inspired from Cuda webinar on reduction kernel04
    * Half the number of threads per block
*/
__global__ void cdist2(const float   * ref,
                        int           ref_nb,
                        const float * query,
                        int           query_nb,
                        int           dim,
                        int           query_index,
                        float       * d_gpu_dist){

    // we need 3 * blockDim * sizeof(float) shared memory
    extern __shared__ float smem[];

    // unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int tid = threadIdx.x;

    // dot product
    smem[tid]                = ((ref[(tid*ref_nb)+blockIdx.x]) * (query[(tid*query_nb)+query_index])) +((ref[((tid+blockDim.x)*ref_nb)+blockIdx.x]) * (query[((tid+blockDim.x)*query_nb)+query_index]));
    // denom_a
    smem[tid+blockDim.x]     = ((ref[(tid*ref_nb)+blockIdx.x]) * (ref[(tid*ref_nb)+blockIdx.x])) + ((ref[((tid+blockDim.x)*ref_nb)+blockIdx.x]) * (ref[((tid+blockDim.x)*ref_nb)+blockIdx.x]));
    // denom_b
    smem[tid+(2*blockDim.x)] = ((query[(tid*query_nb)+query_index]) * (query[(tid*query_nb)+query_index])) + ((query[((tid+blockDim.x)*query_nb)+query_index]) * (query[((tid+blockDim.x)*query_nb)+query_index]));


    __syncthreads();

    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid < s){
            smem[tid] += smem[tid + s];
            smem[tid+blockDim.x] += smem[tid + s + blockDim.x];
            smem[tid+(2*blockDim.x)] += smem[tid + s + (2*blockDim.x)];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0){
        d_gpu_dist[(query_nb*blockIdx.x)+query_index] = smem[0] / (sqrt(smem[blockDim.x]) * sqrt(smem[2*blockDim.x]));
    }
}


// -- CURRENT VERSION --
/*
    * Compute the cosine distance between two vectors
    * inspired from Cuda webinar on reduction kernel04
    * Half the number of threads per block
    * Use padding to handle non Po2 dimensions
    * This version can handle more than 1280 dimensions
*/
__global__ void padded_cdist(const float   * ref,
                        int           ref_nb,
                        const float * query,
                        int           query_nb,
                        int           dim,
                        int           paddedDim,
                        int           query_index,
                        float       * d_gpu_dist){

    // we need 3 * paddedDim * sizeof(float) shared memory
    extern __shared__ float smem[];

    // unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int tid = threadIdx.x;

    // initialize smem, if tid < dim, copy data, else copy 0
    smem[tid] = (tid < dim) ? ((ref[(tid*ref_nb)+blockIdx.x]) * (query[(tid*query_nb)+query_index])) : 0;
    smem[tid+paddedDim] = (tid < dim) ? ((ref[(tid*ref_nb)+blockIdx.x]) * (ref[(tid*ref_nb)+blockIdx.x])) : 0;
    smem[tid+(2*paddedDim)] = (tid < dim) ? ((query[(tid*query_nb)+query_index]) * (query[(tid*query_nb)+query_index])) : 0;

    // perform first reduction step when copying data
    if (tid + blockDim.x < dim){
        smem[tid] += ((ref[((tid+blockDim.x)*ref_nb)+blockIdx.x]) * (query[((tid+blockDim.x)*query_nb)+query_index]));
        smem[tid+paddedDim] += ((ref[((tid+blockDim.x)*ref_nb)+blockIdx.x]) * (ref[((tid+blockDim.x)*ref_nb)+blockIdx.x]));
        smem[tid+(2*paddedDim)] += ((query[((tid+blockDim.x)*query_nb)+query_index]) * (query[((tid+blockDim.x)*query_nb)+query_index]));
    }

    __syncthreads();

    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid < s){
            smem[tid] += smem[tid + s];
            smem[tid+blockDim.x] += smem[tid + s + blockDim.x];
            smem[tid+(2*blockDim.x)] += smem[tid + s + (2*blockDim.x)];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0){
        d_gpu_dist[(query_nb*blockIdx.x)+query_index] = smem[0] / (sqrt(smem[blockDim.x]) * sqrt(smem[2*blockDim.x]));
    }
}


// get min, add delta check how many candidates are in the range
// TODO: do first reduction step when copying data
__global__ void get_min_intrablock(const float* gpu_dist,
                                    int          query_index,
                                    int          query_nb,
                                    float      * min_candidates){

    // set up shared mem
    // blockDim * sizeof(float) for distances and indexes
    extern __shared__ float smem[];
    
    // copy distances and indexes to shared mem
    smem[threadIdx.x] = gpu_dist[(query_nb*blockDim.x*blockIdx.x)+(threadIdx.x*query_nb)+query_index];

    __syncthreads();

    // find min
    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(threadIdx.x < s){
            if(smem[threadIdx.x] > smem[threadIdx.x + s]){
                smem[threadIdx.x] = smem[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (threadIdx.x == 0){
        // printf("min: %f\n", smem[0]);
        min_candidates[blockIdx.x] = smem[0];

        // // print min candidates
        // printf("Block %d\n", blockIdx.x);
        // for(unsigned int i=0; i<blockDim.x; i++){
        //     printf("%f ||", min_candidates[(query_nb*blockIdx.x)+i]);
        // }
        // printf("\n");
    }

}


__global__ void get_min_interblock(const float* min_candidates,
                                    int         query_nb,
                                    float      * min_dist){

    // set up shared mem
    extern __shared__ float smem[];
    
    // copy distances and indexes to shared mem
    smem[threadIdx.x] = min_candidates[threadIdx.x];

    __syncthreads();

    // find min
    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(threadIdx.x < s){
            if(smem[threadIdx.x] > smem[threadIdx.x + s]){
                smem[threadIdx.x] = smem[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (threadIdx.x == 0){
        min_dist[query_nb] = smem[0];
    }
}

// count candidates in range [min, min+delta]
// every thread in the block adds 1 in smem then we sum all vals into global like in "histogram"
__global__ void get_candidates(const float* gpu_dist,
                                const float* min_dist,
                                int          query_index,
                                int          query_nb,
                                float        delta,
                                int        * candidates){
    
    // set up shared mem
    // sizeof(int) for #candidates in the block
    extern __shared__ int smem;

    if(gpu_dist[(query_nb*blockDim.x*blockIdx.x)+(threadIdx.x*query_nb)+query_index] <= min_dist[query_nb] + delta){
        smem++;
    }
                                
}


// Initialize data randomly
void initialize_data(float * ref,
                     int     ref_nb,
                     float * query,
                     int     query_nb,
                     int     dim,
                     int     seed = 51) {

    // Initialize random number generator
    srand(seed); // to get the same results

    // Generate random reference points
    for (int i=0; i<ref_nb*dim; ++i) {
        ref[i] = 10. * (float)(rand() / (double)RAND_MAX);
        // ref[i] = 1;
    }

    // Generate random query points
    for (int i=0; i<query_nb*dim; ++i) {
        query[i] = 10. * (float)(rand() / (double)RAND_MAX);
        // query[i] = 1;
    }
}

// CPU implementation of cosine distance computation
float cosine_distance(const float * ref,
                       int           ref_nb,
                       const float * query,
                       int           query_nb,
                       int           dim,
                       int           ref_index,
                       int           query_index) {

   double dot = 0.0, denom_a = 0.0, denom_b = 0.0 ;
     for(unsigned int d = 0u; d < dim; ++d) {
        dot += ref[d * ref_nb + ref_index] * query[d * query_nb + query_index] ;
        denom_a += ref[d * ref_nb + ref_index] * ref[d * ref_nb + ref_index] ;
        denom_b += query[d * query_nb + query_index] * query[d * query_nb + query_index] ;
    }
    
    return dot / (sqrt(denom_a) * sqrt(denom_b)) ;
    // return dot;
    // return denom_a;
    // return denom_b;
}


void print_matrix(float * matrix, int rows, int cols){
    for(int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){
            printf("%f ||", matrix[i*cols+j]);
        }
        printf("\n");
    }
}

void check_results(float * cpu_dist, float * gpu_dist, int ref_nb, int query_nb, int verbose=0){
    int error = 0;
    for(int i=0; i<ref_nb; ++i) {
        
        for(int j=0; j<query_nb; ++j) {
            if(fabs(gpu_dist[i*query_nb+j] - cpu_dist[i*query_nb+j]) > 0.001){
                if(verbose){
                    printf("Error at index %d, %d\n", i, j);
                    printf("CPU: %f || ", cpu_dist[i*query_nb+j]);
                    printf("GPU: %f\n", gpu_dist[i*query_nb+j]);
                }
                error++;
            }
        }
    }

    printf("Number of errors: %d\n", error);
    printf("Percentage of errors: %f\n", (float) error / (1LL * ref_nb * query_nb) * 100);
}


int main(void) {
    
    // Parameters 0 (to develop your solution)
    const int ref_nb   = 4096;
    const int query_nb = 1024;
    const int dim      = 64;
    const int k        = 16;

    // Parameters 1
    // const int ref_nb   = 16384;
    // const int query_nb = 4096;
    // const int dim      = 128;
    // const int k        = 100;

    // Parameters 2     (not working: too many query & ref points) (splitting in 10 parts make every param a power of 2)
    // const int ref_nb   = 163840;
    // const int query_nb = 40960;
    // const int dim      = 128;
    // const int k        = 16;

    // Parameters 3     
    // const int ref_nb   = 16384;
    // const int query_nb = 4096;
    // const int dim      = 1280;
    // const int k        = 16;

    // Parameters 4
    // const int ref_nb   = 5;
    // const int query_nb = 3;
    // const int dim      = 1280;
    // const int k        = 4;

    // Display
    printf("PARAMETERS\n");
    printf("- Number reference points : %d\n",   ref_nb);
    printf("- Number query points     : %d\n",   query_nb);
    printf("- Dimension of points     : %d\n",   dim);
    printf("- Number of neighbors     : %d\n\n", k);

    // Sanity check
    if (ref_nb<k) {
        printf("Error: k value is larger that the number of reference points\n");
        return EXIT_FAILURE;
    }


    // Allocate input points and output k-NN distances / indexes
    float * ref        = (float*) malloc(ref_nb   * dim * sizeof(float));
    float * query      = (float*) malloc(query_nb * dim * sizeof(float));

    uint64_t o_matrix_size = 1LL * ref_nb * query_nb * sizeof(float);

    float * cpu_dist   = (float*) malloc(o_matrix_size);
    float * h_gpu_dist   = (float*) malloc(o_matrix_size);

    float * knn_dist   = (float*) malloc(o_matrix_size);
    int   * knn_index  = (int*)   malloc(o_matrix_size);

    

    // Allocation checks
    if (!ref || !query || !cpu_dist || !h_gpu_dist || !knn_dist || !knn_index) {
        printf("Error: Memory allocation error\n"); 
        free(ref);
	    free(query);
	    free(cpu_dist);
	    free(h_gpu_dist);
        free(knn_dist);
        free(knn_index);
        return EXIT_FAILURE;
    }

    // Initialize reference and query points with random values
    initialize_data(ref, ref_nb, query, query_nb, dim);

    // COSINE DISTANCE COMPUTATION CPU ----------------------------------------------------------------------------------------------------------------------

    printf("Performing cosine distance computation on CPU\n");

    // start timer
    struct timeval  tv1_cpu, tv2_cpu;
    gettimeofday(&tv1_cpu, NULL);

    // Perform cosine distance computation on CPU
    for(unsigned int query_index=0; query_index<query_nb; query_index++){
        if(query_index % 100 == 0) printf("Query %d\n", query_index);
        for(unsigned int ref_index=0; ref_index<ref_nb; ref_index++){
            cpu_dist[(query_nb*ref_index)+query_index] = cosine_distance(ref, ref_nb, query, query_nb, dim, ref_index, query_index);
        }
    }

    // stop timer
    gettimeofday(&tv2_cpu, NULL);

    // compute and print the elapsed time in millisec
    printf ("Total time = %f milliseconds\n",
             (double) (1000.0 * (tv2_cpu.tv_sec - tv1_cpu.tv_sec) + (tv2_cpu.tv_usec - tv1_cpu.tv_usec) / 1000.0));

    // print results
    // print_matrix(cpu_dist, ref_nb, query_nb);

    // COSINE DISTANCE COMPUTATION GPU ----------------------------------------------------------------------------------------------------------------------
    
    printf("Performing cosine distance computation on GPU\n");

    int blockSize = dim;        // Number of threads per block (this approach cannot handle more than 1024 threads) (last case scenario)
    int gridSize = ref_nb;      // Number of blocks

    printf("blockSize: %d\n", blockSize);
    printf("gridSize: %d\n", gridSize);

    // copy ref and query into cuda mem
    float *d_ref, *d_query;
    float *d_gpu_dist;

    hipMalloc(&d_ref, ref_nb * dim * sizeof(float));
    hipMalloc(&d_query, ref_nb * dim * sizeof(float));

    hipMalloc(&d_gpu_dist, o_matrix_size);
    hipMemset(d_gpu_dist, 100, o_matrix_size);

    hipMemcpy(  d_ref,   ref, ref_nb * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_query, query, ref_nb * dim * sizeof(float), hipMemcpyHostToDevice);


    // start timer 
    struct timeval  tv1, tv2;
    gettimeofday(&tv1, NULL);

    // Calculate the next power of 2 for dim
    int nextPow2 = 1;
    while (nextPow2 < dim) {
        nextPow2 <<= 1;
    }

    // Calculate the number of elements required in smem with padding
    int paddedDim = nextPow2/2;
    int smemSize = 3 * paddedDim * sizeof(float);
    printf("paddedDim: %d\n", paddedDim);

    for(unsigned int query_index=0; query_index<query_nb; query_index++){
        // printf("Query %d\n", query_index);
        padded_cdist<<< gridSize, paddedDim, smemSize >>>(d_ref, ref_nb, d_query, query_nb, dim, paddedDim, query_index, d_gpu_dist);
    }

    // stop timer
    gettimeofday(&tv2, NULL);


    // compute and print the elapsed time in millisec
    printf ("Total time = %f milliseconds\n",
             (double) (1000.0 * (tv2.tv_sec - tv1.tv_sec) + (tv2.tv_usec - tv1.tv_usec) / 1000.0));


    //mem copy back to cpu
    hipMemcpy(h_gpu_dist, d_gpu_dist, o_matrix_size, hipMemcpyDeviceToHost);

    // check results
    check_results(cpu_dist, h_gpu_dist, ref_nb, query_nb);


    // -------------- PART 2: k selection ------------------------------------------------------------------------------------------------------------------------------

    int blockSize2 = 64;        // Test to find the best value theoretically we want second level to use at least 32 blocks to avoid inactive threads in block
    int gridSize2 = ref_nb/blockSize2;

    // allocate cuda mem
    float *d_min_distances;
    float *d_min_dist;

    hipMalloc(&d_min_distances, query_nb * gridSize2 * sizeof(float));
    hipMalloc(&d_min_dist, query_nb * sizeof(float));

    float *gpu_min_dist;
    float *gpu_min_distances;           // it exists for debug purposes

    gpu_min_distances = (float*) malloc(query_nb * gridSize2 * sizeof(float));
    gpu_min_dist = (float*) malloc(query_nb * sizeof(float));
    

    printf("\n\nSearching for min\n");
    printf("blockSize: %d\n", blockSize2);
    printf("gridSize: %d\n", gridSize2);
    // select k nearest neighbors
    for(unsigned int query_index=0; query_index<query_nb; query_index++){
        get_min_intrablock<<< gridSize2, blockSize2, blockSize2 * sizeof(float) >>>(d_gpu_dist, query_index, query_nb, d_min_distances);
        get_min_interblock<<< 1, gridSize2, gridSize2 * sizeof(float) >>>(d_min_distances, query_index, d_min_dist);
    }

    //mem copy back to cpu
    hipMemcpy(gpu_min_dist, d_min_dist, query_nb * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(gpu_min_distances, d_min_distances, query_nb * gridSize2 * sizeof(float), hipMemcpyDeviceToHost);

    // printf("Min distances:\n");
    // for(unsigned int i=0; i<query_nb; i++){
    //     printf("%f ||", gpu_min_dist[i]);
    // }


    // calculate min on cpu ----------------------------------------------------------------------------------------------------------------------
    float * cpu_min_dist = (float*) malloc(query_nb * sizeof(float));

    for(unsigned int query_index=0; query_index<query_nb; query_index++){
        float min = 1;
        for(unsigned int ref_index=0; ref_index<ref_nb; ref_index++){
            if(cpu_dist[(query_nb*ref_index)+query_index] < min){
                min = cpu_dist[(query_nb*ref_index)+query_index];
            }
        }
        // printf("Min: %f\n", min);
        cpu_min_dist[query_index] = min;
    }

    // check results ------------------------------------------------------------------------------------------------------------------------------

    int minfinder_error = 0;

    for(unsigned int i=0; i<query_nb; i++){
        if(fabs(gpu_min_dist[i] - cpu_min_dist[i]) > 0.001){
            printf("Error at index %d\n", i);
            printf("CPU: %f || ", cpu_min_dist[i]);
            printf("GPU: %f\n", gpu_min_dist[i]);

            minfinder_error++;
        }
    }

    printf("Number of errors: %d\n", minfinder_error);
    printf("Percentage of errors: %f\n", (float) minfinder_error / (1LL * query_nb) * 100);


    // -------------- PART 3: add delta and count candidates -------------------------------------------------------------------------------------

    // add delta
    float M = 0.72;         // value selected analyzing the distribution of the min distances
    float delta = 0.1;




    // free cuda mem ------------------------------------------------------------------------------------------------------------------------------
    hipFree(d_ref);
    hipFree(d_query);
    hipFree(d_gpu_dist);
    hipFree(d_min_distances);
    hipFree(d_min_dist);
}