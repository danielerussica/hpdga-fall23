
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>
#include <assert.h>


/*
 * Compute the cosine distance between two vectors
 * inspired from Cuda webinar on reduction kernel03 (mabye extend optimization to kernel04)
*/

__global__ void gpu_get_components(const float * ref,
                                int           ref_nb,
                                const float * query,
                                int           query_nb,
                                int           dim,
                                int           ref_index,
                                int           query_index,
                                float       * d_odot,
                                float       * d_odenom_a,
                                float       * d_odenom_b,
                                float       * d_gpu_dist){
    
    extern __shared__ float smem[];

    // each thread loads one element from global to shared mem (but in this case we need product of corresponding dimension)
    unsigned int tid = threadIdx.x;
    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    // dot product
    smem[tid]                = (ref[(i*ref_nb)+ref_index]) * (query[(i*query_nb)+query_index]);
    // denom_a
    smem[tid+blockDim.x]     = (ref[(i*ref_nb)+ref_index]) * (ref[(i*ref_nb)+ref_index]);
    // denom_b
    smem[tid+(2*blockDim.x)] = (query[(i*query_nb)+query_index]) * (query[(i*query_nb)+query_index]);

    __syncthreads();

    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid < s){
            smem[tid] += smem[tid + s];
            smem[tid+blockDim.x] += smem[tid + s + blockDim.x];
            smem[tid+(2*blockDim.x)] += smem[tid + s + (2*blockDim.x)];
        }
        __syncthreads();
    }
   
    // write result for this block to global memory
    // THIS PART IS NOT NECESSARY IF WORKING WITH ONLY ONE BLOCK
    // if (tid == 0){
    //     d_odot[blockIdx.x]    = smem[0];
    //     d_odenom_a[blockIdx.x] = smem[blockDim.x];
    //     d_odenom_b[blockIdx.x] = smem[2*blockDim.x];
    // } 

    if (tid == 0){
        d_gpu_dist[(query_nb*ref_index)+query_index] = smem[0] / (sqrt(smem[blockDim.x]) * sqrt(smem[2*blockDim.x]));
    }

}

// get idot,idenom_a and idenom_b. Respectively sum them up, and then calculate final result which will be written into gpu_dist
// THIS FUNCTION IS NOT NECESSARY IF WORKING WITH ONLY ONE BLOCK
__global__ void gpu_cosine_distance(int     ref_nb,
                                    int     query_nb,
                                    int     ref_index,
                                    int     query_index,
                                    float * d_idot,
                                    float * d_idenom_a,
                                    float * d_idenom_b,
                                    float * d_gpu_dist){

    extern __shared__ float smem[];

    unsigned int tid = threadIdx.x;
    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    smem[tid] = d_idot[i];
    smem[tid+blockDim.x] = d_idenom_a[i];
    smem[tid+(2*blockDim.x)] = d_idenom_b[i];

    __syncthreads();

    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid < s){
            smem[tid] += smem[tid + s];
            smem[tid+blockDim.x] += smem[tid + s + blockDim.x];
            smem[tid+(2*blockDim.x)] += smem[tid + s + (2*blockDim.x)];
        }
        __syncthreads();
    }

    // write cosine distance result in global memory
    if (tid == 0){
        d_gpu_dist[(query_nb*ref_index)+query_index] = smem[0] / (sqrt(smem[blockDim.x]) * sqrt(smem[2*blockDim.x]));
    }

}

__global__ void cdist(const float * ref,
                        int           ref_nb,
                        const float * query,
                        int           query_nb,
                        int           dim,
                        int           query_index,
                        float       * d_gpu_dist){

    // we need 3 * blockDim * sizeof(float) shared memory
    extern __shared__ float smem[];

    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int tid = threadIdx.x;

    // printf("i: %d\n", i);

    // dot product
    smem[tid]                = (ref[(tid*ref_nb)+blockIdx.x]) * (query[(tid*query_nb)+query_index]);
    // denom_a
    smem[tid+blockDim.x]     = (ref[(tid*ref_nb)+blockIdx.x]) * (ref[(tid*ref_nb)+blockIdx.x]);
    // denom_b
    smem[tid+(2*blockDim.x)] = (query[(tid*query_nb)+query_index]) * (query[(tid*query_nb)+query_index]);



    __syncthreads();

    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid < s){
            smem[tid] += smem[tid + s];
            smem[tid+blockDim.x] += smem[tid + s + blockDim.x];
            smem[tid+(2*blockDim.x)] += smem[tid + s + (2*blockDim.x)];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0){
        d_gpu_dist[(query_nb*blockIdx.x)+query_index] = smem[0] / (sqrt(smem[blockDim.x]) * sqrt(smem[2*blockDim.x]));
    }
}


void initialize_data(float * ref,
                     int     ref_nb,
                     float * query,
                     int     query_nb,
                     int     dim) {

    // Initialize random number generator
    srand(42); // to get the same results

    // Generate random reference points
    for (int i=0; i<ref_nb*dim; ++i) {
        ref[i] = 10. * (float)(rand() / (double)RAND_MAX);
        // ref[i] = 1;
    }

    // Generate random query points
    for (int i=0; i<query_nb*dim; ++i) {
        query[i] = 10. * (float)(rand() / (double)RAND_MAX);
        // query[i] = 1;
    }
}


float cosine_distance(const float * ref,
                       int           ref_nb,
                       const float * query,
                       int           query_nb,
                       int           dim,
                       int           ref_index,
                       int           query_index) {

   double dot = 0.0, denom_a = 0.0, denom_b = 0.0 ;
     for(unsigned int d = 0u; d < dim; ++d) {
        dot += ref[d * ref_nb + ref_index] * query[d * query_nb + query_index] ;
        denom_a += ref[d * ref_nb + ref_index] * ref[d * ref_nb + ref_index] ;
        denom_b += query[d * query_nb + query_index] * query[d * query_nb + query_index] ;
    }
    
    return dot / (sqrt(denom_a) * sqrt(denom_b)) ;
    // return dot;
    // return denom_a;
    // return denom_b;
}



int main(void) {

    
    
    // Parameters 0 (to develop your solution)
    // const int ref_nb   = 4096;
    // const int query_nb = 1024;
    // const int dim      = 64;
    // const int k        = 16;

    // Parameters 1
    const int ref_nb   = 16384;
    const int query_nb = 4096;
    const int dim      = 128;
    const int k        = 100;

    // Parameters 2
    // const int ref_nb   = 163840;
    // const int query_nb = 40960;
    // const int dim      = 128;
    // const int k        = 16;

    // Parameters 3
    // const int ref_nb   = 16384;
    // const int query_nb = 4096;
    // const int dim      = 1280;
    // const int k        = 16;

    // Parameters 4
    // const int ref_nb   = 5;
    // const int query_nb = 3;
    // const int dim      = 128;
    // const int k        = 4;


    int blockSize = dim;        // Number of threads per block (this approach cannot handle more than 1024 threads) (last case scenario)
    int gridSize = ref_nb;      // Number of blocks
    

    // Display
    printf("PARAMETERS\n");
    printf("- Number reference points : %d\n",   ref_nb);
    printf("- Number query points     : %d\n",   query_nb);
    printf("- Dimension of points     : %d\n",   dim);
    printf("- Number of neighbors     : %d\n\n", k);


    // Sanity check
    if (ref_nb<k) {
        printf("Error: k value is larger that the number of reference points\n");
        return EXIT_FAILURE;
    }

    // Allocate input points and output k-NN distances / indexes
    float * ref        = (float*) malloc(ref_nb   * dim * sizeof(float));
    float * query      = (float*) malloc(query_nb * dim * sizeof(float));

    float * cpu_dist   = (float*) malloc(ref_nb * query_nb * sizeof(float));
    float * h_gpu_dist   = (float*) malloc(ref_nb * query_nb * sizeof(float));

    // Allocation checks
    if (!ref || !query || !cpu_dist || !h_gpu_dist) {
        printf("Error: Memory allocation error\n"); 
        free(ref);
	    free(query);
	    free(cpu_dist);
	    free(h_gpu_dist);
        return EXIT_FAILURE;
    }

    // Initialize reference and query points with random values
    initialize_data(ref, ref_nb, query, query_nb, dim);

    printf("Performing cosine distance computation on CPU\n");

    // start timer
    struct timeval  tv1_cpu, tv2_cpu;
    gettimeofday(&tv1_cpu, NULL);

    // Perform cosine distance computation on CPU
    for(int i=0; i<ref_nb; ++i) {
        for(int j=0; j<query_nb; ++j) {
            cpu_dist[i*query_nb+j] = cosine_distance(ref, ref_nb, query, query_nb, dim, i, j);
        }
    }

    // stop timer
    gettimeofday(&tv2_cpu, NULL);

    // compute and print the elapsed time in millisec
    printf ("Total time = %f milliseconds\n",
             (double) (1000.0 * (tv2_cpu.tv_sec - tv1_cpu.tv_sec) + (tv2_cpu.tv_usec - tv1_cpu.tv_usec) / 1000.0));

    // print results
    // for(int i=0; i<ref_nb; ++i) {
    //     for(int j=0; j<query_nb; ++j) {
    //         printf("%f ||", cpu_dist[i*query_nb+j]);
    //     }
    //     printf("\n");
    // }
    
    printf("Performing cosine distance computation on GPU\n");



    printf("blockSize: %d\n", blockSize);
    printf("gridSize: %d\n", gridSize);

    // copy ref and query into cuda mem
    float *d_ref, *d_query;
    float *d_odot, *d_odenom_a, *d_odenom_b;
    float *d_gpu_dist;


    hipMalloc(&d_ref, ref_nb * dim * sizeof(float));
    hipMalloc(&d_query, ref_nb * dim * sizeof(float));

    hipMalloc(&d_odot, gridSize * sizeof(float));
    hipMalloc(&d_odenom_a, gridSize * sizeof(float));
    hipMalloc(&d_odenom_b, gridSize * sizeof(float));

    hipMalloc(&d_gpu_dist, ref_nb * query_nb * sizeof(float));

    // printf("Copying data from host to device\n");
    hipMemcpy(  d_ref,   ref, ref_nb * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_query, query, ref_nb * dim * sizeof(float), hipMemcpyHostToDevice);
    // printf("Done\n\n");

    // start timer
    struct timeval  tv1, tv2;
    gettimeofday(&tv1, NULL);

    // int num_stream = ref_nb;
    // cudaStream_t *streams = (cudaStream_t *) malloc(num_stream * sizeof(cudaStream_t));

    // for(int i=0; i<num_stream; i++){
    //     cudaStreamCreate(&streams[i]);
    // }

    // for(int i=0; i<query_nb; i++){
    //     for(int j=0; j<ref_nb; j++){
    //         gpu_get_components<<< gridSize, blockSize, 3 * blockSize * sizeof(float), streams[j] >>>(d_ref, ref_nb, d_query, query_nb, dim, j, i, d_odot, d_odenom_a, d_odenom_b);            
    //         gpu_cosine_distance<<< 1, gridSize, 3 * gridSize * sizeof(float), streams[j] >>>(ref_nb, query_nb, j, i, d_odot, d_odenom_a, d_odenom_b, d_gpu_dist);
    //     }
    // }


    // for(unsigned int i=0; i<query_nb; i++){
    //     // printf("query %d\n", i);
    //     for(unsigned int j=0; j<ref_nb; j++){
    //         gpu_get_components<<< gridSize, blockSize, 3 * blockSize * sizeof(float) >>>(d_ref, ref_nb, d_query, query_nb, dim, j, i, d_odot, d_odenom_a, d_odenom_b, d_gpu_dist);            
    //         // gpu_cosine_distance<<< 1, gridSize, 3 * gridSize * sizeof(float) >>>(ref_nb, query_nb, j, i, d_odot, d_odenom_a, d_odenom_b, d_gpu_dist);
    //     }
    // }

    for(unsigned int query_index=0; query_index<query_nb; query_index++){
        // printf("Query %d\n", query_index);
        cdist<<< gridSize, blockSize, 3 * blockSize * sizeof(float) >>>(d_ref, ref_nb, d_query, query_nb, dim, query_index, d_gpu_dist);
    }


    // stop timer
    gettimeofday(&tv2, NULL);

    // compute and print the elapsed time in millisec
    printf ("Total time = %f milliseconds\n",
             (double) (1000.0 * (tv2.tv_sec - tv1.tv_sec) + (tv2.tv_usec - tv1.tv_usec) / 1000.0));


    //mem copy back to cpu
    // expensive
    hipMemcpy(h_gpu_dist, d_gpu_dist, ref_nb * query_nb * sizeof(float), hipMemcpyDeviceToHost);

    // print results
            // for(int i=0; i<ref_nb; ++i) {
            //     for(int j=0; j<query_nb; ++j) {
            //         // printf("%f ||",fabs(h_gpu_dist[i*query_nb+j] - cpu_dist[i*query_nb+j]));
            //         printf("%f ||", h_gpu_dist[i*query_nb+j]);
            //     }
            //     printf("\n");
            // }

    // check results
    int error = 0;
    for(int i=0; i<ref_nb; ++i) {
        for(int j=0; j<query_nb; ++j) {
            if(fabs(h_gpu_dist[i*query_nb+j] - cpu_dist[i*query_nb+j]) > 0.001){
                // printf("Error at index (%d, %d)\t", i, j);
                // printf("CPU: %f || GPU: %f\n", cpu_dist[i*query_nb+j], h_gpu_dist[i*query_nb+j]);
                error++;
            }
        }
    }

    printf("Number of errors: %d\n", error);
    printf("Percentage of errors: %f\n", (float) error / (ref_nb * query_nb) * 100);



    // free cuda mem
    hipFree(d_ref);
    hipFree(d_query);
    hipFree(d_odot);
    hipFree(d_odenom_a);
    hipFree(d_odenom_b);
    hipFree(d_gpu_dist);

}