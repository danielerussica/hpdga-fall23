#include "hip/hip_runtime.h"
#include "kernels.cu"
#include "old_kernels.cu"
#include <iostream>


// PROPOSED SOLUTION 1
bool ys_selection_approach(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,    // output fields
                     int *         knn_index) {

    // ds that must be allocated
    float   *d_ref, *d_query;

    hipMalloc(&d_ref, ref_nb * dim * sizeof(float));
    hipMalloc(&d_query, query_nb * dim * sizeof(float));

    hipMemcpy(  d_ref,   ref, ref_nb * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_query, query, query_nb * dim * sizeof(float), hipMemcpyHostToDevice);

    float *d_knn_dist;
    int *d_knn_index;

    hipMalloc(&d_knn_dist, query_nb * k * sizeof(float));
    hipMalloc(&d_knn_index, query_nb * k * sizeof(int));

    // Get device properties
    int deviceId = 0;
    hipSetDevice(deviceId);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);
    // std::cout << "Total GPU Global Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB\n";

    uint64_t o_matrix_size = 1L * ref_nb * query_nb * sizeof(float);

    uint64_t batches = ceil((double) 2*o_matrix_size/(deviceProp.totalGlobalMem));
    // std::cout << "batches = " << o_matrix_size << "/" << deviceProp.totalGlobalMem << " = " << batches << "\n";  

    int gridSize = ref_nb;      // Number of blocks for cdist

    for(unsigned int batch = 0; batch<batches; batch++){
        
        // printf("batch %d\n", batch);

        float   *d_gpu_dist;
        int     *d_index;

        hipMalloc(&d_gpu_dist, o_matrix_size/batches);
        hipMalloc(&d_index, o_matrix_size/batches);

        // Calculate the next power of 2 for dim
        int nextPow2 = 1;
        while (nextPow2 < dim) {
            nextPow2 <<= 1;
        }

        // Calculate the number of elements required in smem with padding
        int paddedDim = nextPow2/2;
        int smemSize = 3 * paddedDim * sizeof(float);

        cdist3<<< gridSize, paddedDim, smemSize >>>(d_ref, ref_nb, d_query, query_nb/batches, batches, dim, paddedDim, batch*(query_nb/batches),d_gpu_dist, d_index);

        // printf("end cdist\n");

        // batch k selection ----------------------------------------------------------------------------------------------------------------------
        int blockSize2 = 1024;
        int gridSize2 = ref_nb/blockSize2;

        nextPow2 = 1;
        while (nextPow2 < gridSize2) {
            nextPow2 <<= 1;
        }

        int interblockGridSize = nextPow2;
        // printf("paddedGridSize2: %d\n", interblockGridSize);

        // allocate cuda mem
        float *d_min_distances;
        int *d_min_indexes;

        hipMalloc(&d_min_distances, query_nb/batches * gridSize2 * sizeof(float));
        hipMalloc(&d_min_indexes,   query_nb/batches * gridSize2 * sizeof(int));

        // for(unsigned int i=0; i<k; i++){
        //     for(unsigned int query_index=0; query_index<query_nb/batches; query_index++){
        //         get_min_intrablock3<<< gridSize2, blockSize2, 2 * blockSize2 * sizeof(float) >>>(d_gpu_dist, d_index, query_index, 0, query_nb/batches, d_min_distances, d_min_indexes);
        //         get_min_interblock5<<< 1, interblockGridSize, 2 * interblockGridSize * sizeof(float) >>>(d_min_distances, d_min_indexes, gridSize2, d_gpu_dist, query_index,  query_nb/batches, batch*(query_nb/batches),batches, i, k, d_knn_dist, d_knn_index);
        //     }
        // }

        for(unsigned int i=0; i<k; i++){
            get_min_intrablock6<<< gridSize2, blockSize2, 2 * blockSize2 * sizeof(float) >>>(d_gpu_dist, d_index, 0, query_nb/batches, d_min_distances, d_min_indexes);
            get_min_interblock6<<< 1, interblockGridSize, 2 * interblockGridSize * sizeof(float) >>>(d_min_distances, d_min_indexes, gridSize2, d_gpu_dist,  query_nb/batches, batch*(query_nb/batches),batches, i, k, d_knn_dist, d_knn_index);
        }

        // printf("end k selection\n");

        hipFree(d_gpu_dist);
        hipFree(d_index);
        hipFree(d_min_distances);
        hipFree(d_min_indexes);
    }



    // mem copy back to cpu
    hipMemcpy(knn_dist, d_knn_dist, query_nb * k * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(knn_index, d_knn_index, query_nb * k * sizeof(int), hipMemcpyDeviceToHost);


    hipFree(d_ref);
    hipFree(d_query);
    hipFree(d_knn_dist);
    hipFree(d_knn_index);

    return true;
}

// PROPOSED SOLUTION 2 (better version)
bool ys_gpu_partial_sort(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,    // output fields
                     int *         knn_index) {

    // ds that must be allocated
    float   *d_ref, *d_query;

    hipMalloc(&d_ref, ref_nb * dim * sizeof(float));
    hipMalloc(&d_query, query_nb * dim * sizeof(float));

    hipMemcpy(  d_ref,   ref, ref_nb * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_query, query, query_nb * dim * sizeof(float), hipMemcpyHostToDevice);

    float *d_knn_dist;
    int *d_knn_index;

    hipMalloc(&d_knn_dist, query_nb * k * sizeof(float));
    hipMalloc(&d_knn_index, query_nb * k * sizeof(int));

    // Get device properties
    int deviceId = 0;
    hipSetDevice(deviceId);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);
    // std::cout << "Total GPU Global Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB\n";

    uint64_t o_matrix_size = 1L * ref_nb * query_nb * sizeof(float);

    uint64_t batches = ceil((double) 2*o_matrix_size/(deviceProp.totalGlobalMem));
    // std::cout << "batches = " << o_matrix_size << "/" << deviceProp.totalGlobalMem << " = " << batches << "\n";  

    int gridSize = ref_nb;      // Number of blocks for cdist

    for(unsigned int batch = 0; batch<batches; batch++){
        
        // printf("batch %d\n", batch);

        float   *d_gpu_dist;
        int     *d_index;

        hipMalloc(&d_gpu_dist, o_matrix_size/batches);
        hipMalloc(&d_index, o_matrix_size/batches);

        // Calculate the next power of 2 for dim
        int nextPow2 = 1;
        while (nextPow2 < dim) {
            nextPow2 <<= 1;
        }

        // Calculate the number of elements required in smem with padding
        int paddedDim = nextPow2/2;
        int smemSize = 3 * paddedDim * sizeof(float);

        cdist3<<< gridSize, paddedDim, smemSize >>>(d_ref, ref_nb, d_query, query_nb/batches, batches, dim, paddedDim, batch*(query_nb/batches), d_gpu_dist, d_index);

        // printf("end cdist\n");

        // batch k selection ----------------------------------------------------------------------------------------------------------------------
        int blockSize2 = 1024;
        int gridSize2 = query_nb/blockSize2;

        gpu_custom_insertion_sort<<< gridSize2, blockSize2>>>(d_gpu_dist, d_index, ref_nb, k, query_nb/batches, 0, d_knn_dist, d_knn_index);

        // printf("end k selection\n");

        hipFree(d_gpu_dist);
        hipFree(d_index);
    }



    // mem copy back to cpu
    hipMemcpy(knn_dist, d_knn_dist, query_nb * k * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(knn_index, d_knn_index, query_nb * k * sizeof(int), hipMemcpyDeviceToHost);


    hipFree(d_ref);
    hipFree(d_query);
    hipFree(d_knn_dist);
    hipFree(d_knn_index);

    return true;
}

// OLD SOLUTION - NOT USED
bool your_solution_only_dist(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,    // output fields
                     int *         knn_index) {

    uint64_t o_matrix_size = 1LL * ref_nb * query_nb * sizeof(float);

    float * cpu_dist   = (float*) malloc(o_matrix_size);
    float * h_gpu_dist = (float*) malloc(o_matrix_size);
    int   * h_gpu_index    = (int*)   malloc(o_matrix_size);

    // int blockSize = dim;        // Number of threads per block
    int gridSize = ref_nb;      // Number of blocks

    // printf("blockSize: %d\n", blockSize);
    // printf("gridSize: %d\n", gridSize);

    // copy ref and query into cuda mem
    float   *d_ref, *d_query;
    float   *d_gpu_dist;
    int     *d_index;

    hipMalloc(&d_ref, ref_nb * dim * sizeof(float));
    hipMalloc(&d_query, query_nb * dim * sizeof(float));

    hipMalloc(&d_gpu_dist, o_matrix_size);
    hipMalloc(&d_index, o_matrix_size);

    hipMemcpy(  d_ref,   ref, ref_nb * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_query, query, query_nb * dim * sizeof(float), hipMemcpyHostToDevice);


    // Calculate the next power of 2 for dim
    int nextPow2 = 1;
    while (nextPow2 < dim) {
        nextPow2 <<= 1;
    }

    // Calculate the number of elements required in smem with padding
    int paddedDim = nextPow2/2;
    int smemSize = 3 * paddedDim * sizeof(float);
    // printf("paddedDim: %d\n", paddedDim);

    for(unsigned int query_index=0; query_index<query_nb; query_index++){
        // printf("Query %d\n", query_index);
        padded_cdist<<< gridSize, paddedDim, smemSize >>>(d_ref, ref_nb, d_query, query_nb, dim, paddedDim, query_index, d_gpu_dist, d_index);
    }

    // mem copy back to cpu
    // very expensive and can be done in parallel with k selection, as long as memcopy is done before actual selection
    hipMemcpy(h_gpu_dist, d_gpu_dist, o_matrix_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_gpu_index, d_index, o_matrix_size, hipMemcpyDeviceToHost);

    hipFree(d_ref);
    hipFree(d_query);


    // K SELECTION ON CPU --------------------------------------------------------------------------------------------------------------------------------

    for(int i=0; i<query_nb; i++){
        insertion_sort_on_matrix(h_gpu_dist, h_gpu_index, ref_nb, k, i, query_nb);

        // Copy k smallest distances and their associated index
        for (int j=0; j<k; ++j) {
        knn_dist[j * query_nb + i]  = h_gpu_dist[(j*query_nb)+i];
        knn_index[j * query_nb + i] = h_gpu_index[(j*query_nb)+i];
        }
    }

   return true;
}


// NOT WORKING 
// solution that compute dists on gpu and compact "sparse array" on gpu using stream compaction
bool your_solution_stream_compaction(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,    // output fields
                     int *         knn_index) {

    uint64_t o_matrix_size = 1LL * ref_nb * query_nb * sizeof(float);

    float * cpu_dist   = (float*) malloc(o_matrix_size);
    float * h_gpu_dist = (float*) malloc(o_matrix_size);
    int   * h_gpu_index    = (int*)   malloc(o_matrix_size);
    int  * h_gpu_prefix_sum    = (int*)   malloc(o_matrix_size);

    // int blockSize = dim;        // Number of threads per block
    int gridSize = ref_nb;      // Number of blocks

    // printf("blockSize: %d\n", blockSize);
    // printf("gridSize: %d\n", gridSize);

    // copy ref and query into cuda mem
    float   *d_ref, *d_query;
    float   *d_gpu_dist;
    int     *d_index;
    int     *d_valid;
    int     *d_prefix_sum;

    hipMalloc(&d_ref, ref_nb * dim * sizeof(float));
    hipMalloc(&d_query, query_nb * dim * sizeof(float));

    hipMalloc(&d_gpu_dist, o_matrix_size);
    hipMalloc(&d_index, o_matrix_size);
    hipMalloc(&d_valid, o_matrix_size);
    hipMalloc(&d_prefix_sum, query_nb * sizeof(int));

    hipMemcpy(  d_ref,   ref, ref_nb * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_query, query, query_nb * dim * sizeof(float), hipMemcpyHostToDevice);


    // Calculate the next power of 2 for dim
    int nextPow2 = 1;
    while (nextPow2 < dim) {
        nextPow2 <<= 1;
    }

    // Calculate the number of elements required in smem with padding
    int paddedDim = nextPow2/2;
    int smemSize = 3 * paddedDim * sizeof(float);
    // printf("paddedDim: %d\n", paddedDim);

    for(unsigned int query_index=0; query_index<query_nb; query_index++){
        // printf("Query %d\n", query_index);
        padded_cdist_with_valid<<< gridSize, paddedDim, smemSize >>>(d_ref, ref_nb, d_query, query_nb, dim, paddedDim, query_index, d_gpu_dist, d_index, d_valid);
    }

    hipFree(d_ref);
    hipFree(d_query);

    int blockSize_prefix_sum = 1024;

    // stream compaction
    for(unsigned int query_index=0; query_index<query_nb; query_index++){
        prefix_sum<<< ref_nb/blockSize_prefix_sum, blockSize_prefix_sum, blockSize_prefix_sum * sizeof(int) >>>(d_valid, query_nb, query_index, d_prefix_sum);
    }

    // print out prefix sum in a file
    // hipMemcpy(h_gpu_prefix_sum, d_valid, o_matrix_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_gpu_prefix_sum, d_prefix_sum, o_matrix_size, hipMemcpyDeviceToHost);

    FILE *f = fopen("prefix_sum.txt", "w");
    if (f == NULL)
    {
        printf("Error opening file!\n");
        exit(1);
    }

    for(unsigned int i=0; i<10; i++){
        for(unsigned int j=0; j<ref_nb; j++){
            fprintf(f, "%d ", h_gpu_prefix_sum[(query_nb*j)+i]);
        }
        fprintf(f, "\n");
    }

    fclose(f);

    

   return true;
}

// OLD SOLUTION - NOT USED
bool your_solution_pick_k_on_gpu(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,    // output fields
                     int *         knn_index) {



    // Solution that naively pick k-smallest on gpu using reduction
    uint64_t o_matrix_size = 1LL * ref_nb * query_nb * sizeof(float);

    float * cpu_dist   = (float*) malloc(o_matrix_size);
    float * h_gpu_dist = (float*) malloc(o_matrix_size);
    int   * h_gpu_index    = (int*)   malloc(o_matrix_size);

    // int blockSize = dim;        // Number of threads per block
    int gridSize = ref_nb;      // Number of blocks

    // printf("blockSize: %d\n", blockSize);
    // printf("gridSize: %d\n", gridSize);

    // copy ref and query into cuda mem
    float   *d_ref, *d_query;
    float   *d_gpu_dist;
    int     *d_index;

    hipMalloc(&d_ref, ref_nb * dim * sizeof(float));
    hipMalloc(&d_query, query_nb * dim * sizeof(float));

    hipMalloc(&d_gpu_dist, o_matrix_size);
    hipMalloc(&d_index, o_matrix_size);

    hipMemcpy(  d_ref,   ref, ref_nb * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_query, query, query_nb * dim * sizeof(float), hipMemcpyHostToDevice);


    // Calculate the next power of 2 for dim
    int nextPow2 = 1;
    while (nextPow2 < dim) {
        nextPow2 <<= 1;
    }

    // Calculate the number of elements required in smem with padding
    int paddedDim = nextPow2/2;
    int smemSize = 3 * paddedDim * sizeof(float);
    // printf("paddedDim: %d\n", paddedDim);

    for(unsigned int query_index=0; query_index<query_nb; query_index++){
        // printf("Query %d\n", query_index);
        padded_cdist<<< gridSize, paddedDim, smemSize >>>(d_ref, ref_nb, d_query, query_nb, dim, paddedDim, query_index, d_gpu_dist, d_index);
    }

    hipFree(d_ref);
    hipFree(d_query);


    // K SELECTION --------------------------------------------------------------------------------------------------------------------------------

    int blockSize2 = 64;        // Test to find the best value theoretically we want second level to use at least 32 blocks to avoid inactive threads in block
    int gridSize2 = ref_nb/blockSize2;

    // allocate cuda mem
    float *d_min_distances;
    int *d_min_indexes;

    float *d_knn_dist;
    int *d_knn_index;
    

    hipMalloc(&d_min_distances, query_nb * gridSize2 * sizeof(float));
    hipMalloc(&d_min_indexes, query_nb * gridSize2 * sizeof(int));

    hipMalloc(&d_knn_dist, query_nb * k * sizeof(float));
    hipMalloc(&d_knn_index, query_nb * k * sizeof(int));
    
    // to do: try to implement stream version of this
    for(unsigned int i=0; i<k; i++){
        for(unsigned int query_index=0; query_index<query_nb; query_index++){
            get_min_intrablock2<<< gridSize2, blockSize2, 2 * blockSize2 * sizeof(float) >>>(d_gpu_dist, d_index, query_index, query_nb, d_min_distances, d_min_indexes);
            get_min_interblock2<<< 1, gridSize2, 2 * gridSize2 * sizeof(float) >>>(d_min_distances, d_min_indexes, d_gpu_dist, query_index, query_nb, i, k, d_knn_dist, d_knn_index);
        }
    }

    // mem copy back to cpu
    hipMemcpy(knn_dist, d_knn_dist, query_nb * k * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(knn_index, d_knn_index, query_nb * k * sizeof(int), hipMemcpyDeviceToHost);

    return true;
}


// OLD SOLUTION - NOT USED
bool your_solution_pick_k_on_gpu_w_stream(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,    // output fields
                     int *         knn_index) {



    // Solution that naively pick k-smallest on gpu using reduction
    uint64_t o_matrix_size = 1LL * ref_nb * query_nb * sizeof(float);

    float * cpu_dist   = (float*) malloc(o_matrix_size);
    float * h_gpu_dist = (float*) malloc(o_matrix_size);
    int   * h_gpu_index    = (int*)   malloc(o_matrix_size);

    // int blockSize = dim;        // Number of threads per block
    int gridSize = ref_nb;      // Number of blocks

    // printf("blockSize: %d\n", blockSize);
    // printf("gridSize: %d\n", gridSize);

    // copy ref and query into cuda mem
    float   *d_ref, *d_query;
    float   *d_gpu_dist;
    int     *d_index;

    hipMalloc(&d_ref, ref_nb * dim * sizeof(float));
    hipMalloc(&d_query, query_nb * dim * sizeof(float));

    hipMalloc(&d_gpu_dist, o_matrix_size);
    hipMalloc(&d_index, o_matrix_size);

    hipMemcpy(  d_ref,   ref, ref_nb * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_query, query, query_nb * dim * sizeof(float), hipMemcpyHostToDevice);


    // COSINE DISTANCE --------------------------------------------------------------------------------------------------------------------------------

    // Calculate the next power of 2 for dim
    int nextPow2 = 1;
    while (nextPow2 < dim) {
        nextPow2 <<= 1;
    }

    // Calculate the number of elements required in smem with padding
    int paddedDim = nextPow2/2;
    int smemSize = 3 * paddedDim * sizeof(float);
    // printf("paddedDim: %d\n", paddedDim);

    for(unsigned int query_index=0; query_index<query_nb; query_index++){
        // printf("Query %d\n", query_index);
        padded_cdist<<< gridSize, paddedDim, smemSize >>>(d_ref, ref_nb, d_query, query_nb, dim, paddedDim, query_index, d_gpu_dist, d_index);
    }

    hipDeviceSynchronize();


    // K SELECTION --------------------------------------------------------------------------------------------------------------------------------

    int blockSize2 = 64;        // (tested: 64>512 with param1) Test to find the best value theoretically we want second level to use at least 32 blocks to avoid inactive threads in block
    int gridSize2 = ref_nb/blockSize2;

    // allocate cuda mem
    float *d_min_distances;
    int *d_min_indexes;

    float *d_knn_dist;
    int *d_knn_index;
    

    hipMalloc(&d_min_distances, query_nb * gridSize2 * sizeof(float));
    hipMalloc(&d_min_indexes, query_nb * gridSize2 * sizeof(int));

    hipMalloc(&d_knn_dist, query_nb * k * sizeof(float));
    hipMalloc(&d_knn_index, query_nb * k * sizeof(int));
    
    // create n streams, divide gpu_dist and gpu_index into n parts and do k selection on each stream
    int n_streams = 128;
    hipStream_t stream[n_streams];

    for (int i = 0; i < n_streams; i++) {
        hipStreamCreate(&stream[i]);
    }

    // array of streams

    int offset = query_nb/n_streams;

    for(unsigned int i=0; i<k; i++){
        for(unsigned int query_index=0; query_index<query_nb/n_streams; query_index++){
            for(int j=0; j<n_streams; j++){
                
                get_min_intrablock3<<< gridSize2, blockSize2, 2 * blockSize2 * sizeof(float), stream[j] >>>(d_gpu_dist, d_index, query_index, offset*j, query_nb, d_min_distances, d_min_indexes);
                get_min_interblock3<<< 1, gridSize2, 2 * gridSize2 * sizeof(float), stream[j] >>>(d_min_distances, d_min_indexes, d_gpu_dist, query_index, offset*j, query_nb, i, k, d_knn_dist, d_knn_index);
            }
        }
    }

    // mem copy back to cpu
    hipMemcpy(knn_dist, d_knn_dist, query_nb * k * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(knn_index, d_knn_index, query_nb * k * sizeof(int), hipMemcpyDeviceToHost);


    // //fprintf knn_dist and knn_index to file
    // FILE *f = fopen("knn_dist.txt", "w");
    // if (f == NULL)
    // {
    //     printf("Error opening file!\n");
    //     exit(1);
    // }

    // for(unsigned int i=0; i<query_nb; i++){
    //     for(unsigned int j=0; j<k; j++){
    //         fprintf(f, "%f ", knn_dist[(query_nb*j)+i]);
    //     }
    //     fprintf(f, "\n");
    // }

    // fclose(f);

    // f = fopen("knn_index.txt", "w");
    // if (f == NULL)
    // {
    //     printf("Error opening file!\n");
    //     exit(1);
    // }

    // for(unsigned int i=0; i<query_nb; i++){
    //     for(unsigned int j=0; j<k; j++){
    //         fprintf(f, "%d ", knn_index[(query_nb*j)+i]);
    //     }
    //     fprintf(f, "\n");
    // }

    // fclose(f);

    // free cuda mem
    hipFree(d_ref);
    hipFree(d_query);
    hipFree(d_min_distances);
    hipFree(d_min_indexes);
    hipFree(d_knn_dist);
    hipFree(d_knn_index);
    hipFree(d_gpu_dist);
    hipFree(d_index);



    return true;
}

// OLD SOLUTION - NOT USED
bool ys_pick_kgpu_innerfor(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,    // output fields
                     int *         knn_index) {



    // Solution that naively pick k-smallest on gpu using reduction
    uint64_t o_matrix_size = 1LL * ref_nb * query_nb * sizeof(float);

    float * cpu_dist   = (float*) malloc(o_matrix_size);
    float * h_gpu_dist = (float*) malloc(o_matrix_size);
    int   * h_gpu_index    = (int*)   malloc(o_matrix_size);

    // int blockSize = dim;        // Number of threads per block
    int gridSize = ref_nb;      // Number of blocks

    // printf("blockSize: %d\n", blockSize);
    // printf("gridSize: %d\n", gridSize);

    // copy ref and query into cuda mem
    float   *d_ref, *d_query;
    float   *d_gpu_dist;
    int     *d_index;

    hipMalloc(&d_ref, ref_nb * dim * sizeof(float));
    hipMalloc(&d_query, query_nb * dim * sizeof(float));

    hipMalloc(&d_gpu_dist, o_matrix_size);
    hipMalloc(&d_index, o_matrix_size);

    hipMemcpy(  d_ref,   ref, ref_nb * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_query, query, query_nb * dim * sizeof(float), hipMemcpyHostToDevice);


    // Calculate the next power of 2 for dim
    int nextPow2 = 1;
    while (nextPow2 < dim) {
        nextPow2 <<= 1;
    }

    // Calculate the number of elements required in smem with padding
    int paddedDim = nextPow2/2;
    int smemSize = 3 * paddedDim * sizeof(float);
    // printf("paddedDim: %d\n", paddedDim);

    padded_cdist2_innerfor<<< gridSize, paddedDim, smemSize >>>(d_ref, ref_nb, d_query, query_nb, dim, paddedDim, d_gpu_dist, d_index);

    hipFree(d_ref);
    hipFree(d_query);


    // K SELECTION --------------------------------------------------------------------------------------------------------------------------------

    int blockSize2 = int(sqrt(double(ref_nb)));
    int gridSize2 = blockSize2;

    // printf("blockSize: %d\n", blockSize2);

    // allocate cuda mem
    float *d_min_distances;
    int *d_min_indexes;

    float *d_knn_dist;
    int *d_knn_index;
    

    hipMalloc(&d_min_distances, query_nb * gridSize2 * sizeof(float));
    hipMalloc(&d_min_indexes, query_nb * gridSize2 * sizeof(int));

    hipMalloc(&d_knn_dist, query_nb * k * sizeof(float));
    hipMalloc(&d_knn_index, query_nb * k * sizeof(int));
    
    for(unsigned int i=0; i<k; i++){
        for(unsigned int query_index=0; query_index<query_nb; query_index++){
            get_min4<<< blockSize2, gridSize2, 2 * gridSize2 * sizeof(float) >>>(d_gpu_dist, d_index, query_index, query_nb, d_min_distances, d_min_indexes, i, d_knn_dist, d_knn_index);
        }
    }


    


    // mem copy back to cpu
    hipMemcpy(knn_dist, d_knn_dist, query_nb * k * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(knn_index, d_knn_index, query_nb * k * sizeof(int), hipMemcpyDeviceToHost);



    // //fprintf knn_dist and knn_index to file
    FILE *f = fopen("knn_dist.txt", "w");
    if (f == NULL)
    {
        printf("Error opening file!\n");
        exit(1);
    }

    for(unsigned int i=0; i<query_nb; i++){
        for(unsigned int j=0; j<k; j++){
            fprintf(f, "%f ", knn_dist[(query_nb*j)+i]);
        }
        fprintf(f, "\n");
    }

    fclose(f);

    f = fopen("knn_index.txt", "w");
    if (f == NULL)
    {
        printf("Error opening file!\n");
        exit(1);
    }

    for(unsigned int i=0; i<query_nb; i++){
        for(unsigned int j=0; j<k; j++){
            fprintf(f, "%d ", knn_index[(query_nb*j)+i]);
        }
        fprintf(f, "\n");
    }

    fclose(f);

    return true;
}


