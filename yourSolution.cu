#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>


// -- OLD VERSION --
/*
    * Compute the cosine distance between two vectors
    * inspired from Cuda webinar on reduction kernel03 (mabye extend optimization to kernel04)
    * In this version, each block computes a single cosine distance between a variable ref point vs a query point, each block has "dim" threads
*/
__global__ void cdist(const float   * ref,
                        int           ref_nb,
                        const float * query,
                        int           query_nb,
                        int           dim,
                        int           query_index,
                        float       * d_gpu_dist){

    // we need 3 * blockDim * sizeof(float) shared memory
    extern __shared__ float smem[];

    // unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int tid = threadIdx.x;

    // dot product
    smem[tid]                = ((ref[(tid*ref_nb)+blockIdx.x]) * (query[(tid*query_nb)+query_index])) ;
    // denom_a
    smem[tid+blockDim.x]     = (ref[(tid*ref_nb)+blockIdx.x]) * (ref[(tid*ref_nb)+blockIdx.x]) ;
    // denom_b
    smem[tid+(2*blockDim.x)] = (query[(tid*query_nb)+query_index]) * (query[(tid*query_nb)+query_index]) ;

    if(smem[tid] == 0){
        printf("smem[%d]: %f\n", tid, smem[tid]);
        printf("smem[%d]: %f\n", tid+blockDim.x, smem[tid+blockDim.x]);
        printf("smem[%d]: %f\n", tid+(2*blockDim.x), smem[tid+(2*blockDim.x)]);
    }


    __syncthreads();

    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid < s){
            smem[tid] += smem[tid + s];
            smem[tid+blockDim.x] += smem[tid + s + blockDim.x];
            smem[tid+(2*blockDim.x)] += smem[tid + s + (2*blockDim.x)];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0){
        d_gpu_dist[(query_nb*blockIdx.x)+query_index] = smem[0] / (sqrt(smem[blockDim.x]) * sqrt(smem[2*blockDim.x]));
    }
}


// -- OLD VERSION --
/*
    * Compute the cosine distance between two vectors
    * inspired from Cuda webinar on reduction kernel04
    * Half the number of threads per block
*/
__global__ void cdist2(const float   * ref,
                        int           ref_nb,
                        const float * query,
                        int           query_nb,
                        int           dim,
                        int           query_index,
                        float       * d_gpu_dist){

    // we need 3 * blockDim * sizeof(float) shared memory
    extern __shared__ float smem[];

    // unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int tid = threadIdx.x;

    // dot product
    smem[tid]                = ((ref[(tid*ref_nb)+blockIdx.x]) * (query[(tid*query_nb)+query_index])) +((ref[((tid+blockDim.x)*ref_nb)+blockIdx.x]) * (query[((tid+blockDim.x)*query_nb)+query_index]));
    // denom_a
    smem[tid+blockDim.x]     = ((ref[(tid*ref_nb)+blockIdx.x]) * (ref[(tid*ref_nb)+blockIdx.x])) + ((ref[((tid+blockDim.x)*ref_nb)+blockIdx.x]) * (ref[((tid+blockDim.x)*ref_nb)+blockIdx.x]));
    // denom_b
    smem[tid+(2*blockDim.x)] = ((query[(tid*query_nb)+query_index]) * (query[(tid*query_nb)+query_index])) + ((query[((tid+blockDim.x)*query_nb)+query_index]) * (query[((tid+blockDim.x)*query_nb)+query_index]));


    __syncthreads();

    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid < s){
            smem[tid] += smem[tid + s];
            smem[tid+blockDim.x] += smem[tid + s + blockDim.x];
            smem[tid+(2*blockDim.x)] += smem[tid + s + (2*blockDim.x)];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0){
        d_gpu_dist[(query_nb*blockIdx.x)+query_index] = smem[0] / (sqrt(smem[blockDim.x]) * sqrt(smem[2*blockDim.x]));
    }
}


// -- CURRENT VERSION --
/*
    * Compute the cosine distance between two vectors
    * inspired from Cuda webinar on reduction kernel04
    * Half the number of threads per block compared to previous version
    * Use padding to handle non Po2 dimensions
    * This version can handle more than 1280 dimensions (max 2048)
*/
__global__ void padded_cdist(const float   * ref,
                        int           ref_nb,
                        const float * query,
                        int           query_nb,
                        int           dim,
                        int           paddedDim,
                        int           query_index,
                        float       * d_gpu_dist,
                        int         * d_index){

    // we need 3 * paddedDim * sizeof(float) shared memory
    extern __shared__ float smem[];

    // unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int tid = threadIdx.x;

    // initialize smem, if tid < dim, copy data, else copy 0
    smem[tid]               = (tid < dim) ? ((ref[(tid*ref_nb)+blockIdx.x]) * (query[(tid*query_nb)+query_index]))      : 0;
    smem[tid+paddedDim]     = (tid < dim) ? ((ref[(tid*ref_nb)+blockIdx.x]) * (ref[(tid*ref_nb)+blockIdx.x]))           : 0;
    smem[tid+(2*paddedDim)] = (tid < dim) ? ((query[(tid*query_nb)+query_index]) * (query[(tid*query_nb)+query_index])) : 0;

    // perform first reduction step when copying data
    if (tid + blockDim.x < dim){
        smem[tid]               += ((ref[((tid+blockDim.x)*ref_nb)+blockIdx.x]) * (query[((tid+blockDim.x)*query_nb)+query_index]));
        smem[tid+paddedDim]     += ((ref[((tid+blockDim.x)*ref_nb)+blockIdx.x]) * (ref[((tid+blockDim.x)*ref_nb)+blockIdx.x]));
        smem[tid+(2*paddedDim)] += ((query[((tid+blockDim.x)*query_nb)+query_index]) * (query[((tid+blockDim.x)*query_nb)+query_index]));
    }

    __syncthreads();

    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid < s){
            smem[tid]                += smem[tid + s];
            smem[tid+blockDim.x]     += smem[tid + s + blockDim.x];
            smem[tid+(2*blockDim.x)] += smem[tid + s + (2*blockDim.x)];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0){
        d_gpu_dist[(query_nb*blockIdx.x)+query_index] = smem[0] / (sqrt(smem[blockDim.x]) * sqrt(smem[2*blockDim.x]));
        d_index[(query_nb*blockIdx.x)+query_index] = blockIdx.x;
    }
}


// get min, add delta check how many candidates are in the range
// TODO: do first reduction step when copying data
__global__ void get_min_intrablock(const float* gpu_dist,
                                    int          query_index,
                                    int          query_nb,
                                    float      * min_candidates){

    // set up shared mem
    // blockDim * sizeof(float) for distances and indexes
    extern __shared__ float smem[];
    
    // copy distances and indexes to shared mem
    smem[threadIdx.x] = gpu_dist[(query_nb*blockDim.x*blockIdx.x)+(threadIdx.x*query_nb)+query_index];

    __syncthreads();

    // find min
    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(threadIdx.x < s){
            if(smem[threadIdx.x] > smem[threadIdx.x + s]){
                smem[threadIdx.x] = smem[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (threadIdx.x == 0){
        // printf("min: %f\n", smem[0]);
        min_candidates[blockIdx.x] = smem[0];

        // // print min candidates
        // printf("Block %d\n", blockIdx.x);
        // for(unsigned int i=0; i<blockDim.x; i++){
        //     printf("%f ||", min_candidates[(query_nb*blockIdx.x)+i]);
        // }
        // printf("\n");
    }

}


__global__ void get_min_interblock(const float* min_candidates,
                                    int         query_nb,
                                    float      * min_dist){

    // set up shared mem
    extern __shared__ float smem[];
    
    // copy distances and indexes to shared mem
    smem[threadIdx.x] = min_candidates[threadIdx.x];

    __syncthreads();

    // find min
    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(threadIdx.x < s){
            if(smem[threadIdx.x] > smem[threadIdx.x + s]){
                smem[threadIdx.x] = smem[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (threadIdx.x == 0){
        min_dist[query_nb] = smem[0];
    }
}

// count candidates in range [min, min+delta]
// every thread in the block adds 1 in smem then we sum all vals into global like in "histogram"
// do it using 1 block, each thread handles more queries in a for loop
__global__ void get_candidates(const float* gpu_dist,
                                const float* min_dist,
                                int          query_index,
                                int          ref_nb,
                                int          query_nb,
                                float        delta,
                                int        * candidates,
                                int        * count){

    // count candidates, every thread handles more queries and if condition is true flags corresponding cell and increase counter
    for(unsigned int i=0; i<ref_nb/blockDim.x; i++){
        if(gpu_dist[(query_nb*blockDim.x*i)+(threadIdx.x*query_nb)+query_index] < min_dist[query_index] + delta){
            candidates[(query_nb*blockDim.x*i)+(threadIdx.x*query_nb)+query_index] = 1;
            // printf("adding");
            atomicAdd(&count[query_index], 1);
        }
    }

}

// check if we have enough candidates in range [min, min+delta] for each query
__global__ void check_min_k(const int * count,
                            int       k,
                            int     * flag){
    
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if(count[i] < k){
        *flag = 1;
    }


}

/**
 * Sort the distances stopping at K sorted elements
 * exploit mask: h_candidates
 * NOT WORKING
 */
void masked_insertion_sort(float *dist, int *index, int * mask, int length, int k, int query_index, int query_nb){
    // Initialize the first index
    index[0] = 0;

    int skip_counter = 0;

    // Go through all points
    for(unsigned int i=0; i<length; i++){

        float curr_dist = dist[i]; 
        int curr_index = index[i];

        // mask is a matrix of size query_nb * ref_nb
        if(mask[(i*query_nb)+query_index] == 0){
            skip_counter++;
            // printf("Skipping %d, ", i);
            continue;
        }

        // Skip the current value if its index is >= k and if it's higher the k-th already sorted smallest value
        if (i >= k && curr_dist >= dist[k-1]) {
            // printf("Skipping %d, ", i);
            continue;
        }

        

        // Shift values (and indexes) higher that the current distance to the right
        int j = min(i-skip_counter, k-1);
        while (j > 0 && dist[j-1] > curr_dist) {
            // printf("Shifting %d to %d\n", j-1, j);
            dist[j]  = dist[j-1];
            index[j] = index[j-1];
            --j;
        }

        // Write the current distance and index at their position
        dist[j]  = curr_dist;
        index[j] = curr_index; 

        // printf("passing %d\n", i);
    }
}


/**
 * Selection sort the distances stopping at K sorted elements
 * exploit mask: h_candidates
 */
void selection_sort(float *dist, int *index, int *mask, int length, int k, int query_index, int query_nb, float *knn_dist, int *knn_index){

    for(int i=0; i<k; i++){
        float min_value = 1;
        int min_index = 0;

        for(int j=0; j<length; j++){
            if(mask[(j*query_nb)+query_index] == 0){
                continue;
            }
            if(dist[(j*query_nb)+query_index] < min_value){
                min_value = dist[(j*query_nb)+query_index];
                min_index = index[(j*query_nb)+query_index];
            }
        }

        // place values in knn_dist and knn_index
        knn_dist[(query_nb*i)+query_index] = min_value;
        knn_index[(query_nb*i)+query_index] = min_index;

        dist[(min_index*query_nb)+query_index] = 1;

    }
}


/**
 * Sort the distances stopping at K sorted elements
 */
void  insertion_sort_on_matrix(float *dist, int *index, int length, int k, int query_index, int query_nb){

    // Initialise the first index
    index[0] = 0;

    // Go through all points
    for (int i=1; i<length; ++i) {

        // Store current distance and associated index
        float curr_dist  = dist[(i*query_nb)+query_index];
        int   curr_index = i;

        // Skip the current value if its index is >= k and if it's higher the k-th slready sorted mallest value
        if (i >= k && curr_dist >= dist[(k-1)*query_nb+query_index]) {
            continue;
        }

        // Shift values (and indexes) higher that the current distance to the right
        int j = min(i, k-1);
        while (j > 0 && dist[(j-1)*query_nb+query_index] > curr_dist) {
            dist[(j*query_nb)+query_index]  = dist[((j-1)*query_nb)+query_index];
            index[(j*query_nb)+query_index] = index[((j-1)*query_nb)+query_index];
            --j;
        }

        // Write the current distance and index at their position
        dist[(j*query_nb)+query_index]  = curr_dist;
        index[(j*query_nb)+query_index] = curr_index; 
    }
}


bool your_solution_baseline(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,    // output fields
                     int *         knn_index) {

    uint64_t o_matrix_size = 1LL * ref_nb * query_nb * sizeof(float);

    float * cpu_dist   = (float*) malloc(o_matrix_size);
    float * h_gpu_dist = (float*) malloc(o_matrix_size);
    int   * h_gpu_index    = (int*)   malloc(o_matrix_size);

    int blockSize = dim;        // Number of threads per block
    int gridSize = ref_nb;      // Number of blocks

    // printf("blockSize: %d\n", blockSize);
    // printf("gridSize: %d\n", gridSize);

    // copy ref and query into cuda mem
    float   *d_ref, *d_query;
    float   *d_gpu_dist;
    int     *d_index;

    hipMalloc(&d_ref, ref_nb * dim * sizeof(float));
    hipMalloc(&d_query, ref_nb * dim * sizeof(float));

    hipMalloc(&d_gpu_dist, o_matrix_size);
    hipMalloc(&d_index, o_matrix_size);

    hipMemcpy(  d_ref,   ref, ref_nb * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_query, query, ref_nb * dim * sizeof(float), hipMemcpyHostToDevice);


    // Calculate the next power of 2 for dim
    int nextPow2 = 1;
    while (nextPow2 < dim) {
        nextPow2 <<= 1;
    }

    // Calculate the number of elements required in smem with padding
    int paddedDim = nextPow2/2;
    int smemSize = 3 * paddedDim * sizeof(float);
    // printf("paddedDim: %d\n", paddedDim);

    for(unsigned int query_index=0; query_index<query_nb; query_index++){
        // printf("Query %d\n", query_index);
        padded_cdist<<< gridSize, paddedDim, smemSize >>>(d_ref, ref_nb, d_query, query_nb, dim, paddedDim, query_index, d_gpu_dist, d_index);
    }

    // mem copy back to cpu
    // very expensive and can be done in parallel with k selection, as long as memcopy is done before actual selection
    hipMemcpy(h_gpu_dist, d_gpu_dist, o_matrix_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_gpu_index, d_index, o_matrix_size, hipMemcpyDeviceToHost);

    hipFree(d_ref);
    hipFree(d_query);


    // K SELECTION --------------------------------------------------------------------------------------------------------------------------------

    int blockSize2 = 64;        // Test to find the best value theoretically we want second level to use at least 32 blocks to avoid inactive threads in block
    int gridSize2 = ref_nb/blockSize2;

    // allocate cuda mem
    float *d_min_distances;
    float *d_min_dist;

    hipMalloc(&d_min_distances, query_nb * gridSize2 * sizeof(float));
    hipMalloc(&d_min_dist, query_nb * sizeof(float));
    

    // printf("\n\nSearching for min\n");
    // printf("blockSize: %d\n", blockSize2);
    // printf("gridSize: %d\n", gridSize2);
    // get min for each query
    for(unsigned int query_index=0; query_index<query_nb; query_index++){
        get_min_intrablock<<< gridSize2, blockSize2, blockSize2 * sizeof(float) >>>(d_gpu_dist, query_index, query_nb, d_min_distances);
        get_min_interblock<<< 1, gridSize2, gridSize2 * sizeof(float) >>>(d_min_distances, query_index, d_min_dist);
    }

    // -------------- PART 2.5: add delta and count candidates -------------------------------------------------------------------------------------

    // add delta
    float delta = 0.1;

    int *d_candidates;
    int *d_count;
    int *d_flag;

    hipMalloc(&d_candidates, query_nb * ref_nb * sizeof(float));
    hipMalloc(&d_count, query_nb * sizeof(int));

    hipMemset(d_candidates, 0, query_nb * ref_nb * sizeof(float));
    hipMemset(d_count, 0, query_nb * sizeof(int));

    int *h_count = (int*) malloc(query_nb * sizeof(int));
    int *h_candidates = (int*) calloc(query_nb * ref_nb, sizeof(int));
    int *h_flag = (int*) malloc(sizeof(int));

    int blockSize3 = 1024;
    int gridSize3 = query_nb/blockSize3;       

    int collected_candidates = 0;
    while(!collected_candidates){
        // printf("\n\nSearching for candidates\n");
        for(unsigned int query_index=0; query_index<query_nb; query_index++){
            get_candidates<<< 1, blockSize2 >>> (d_gpu_dist, d_min_dist, query_index, ref_nb, query_nb, delta, d_candidates, d_count);
        }

        hipDeviceSynchronize();

        check_min_k<<<gridSize3, blockSize3>>>(d_count, k, d_flag);


        *h_flag = 0;
        hipMemcpy(h_flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);

        if(*h_flag == 0){
            collected_candidates = 1;
        }
        else{
            delta += delta/2;
            hipMemset(d_count, 0, query_nb * sizeof(int));
        }

    }

    hipMemcpy(h_count, d_count, query_nb * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_candidates, d_candidates, query_nb * ref_nb * sizeof(int), hipMemcpyDeviceToHost);


    // printf("number of candidates:\n");
    // for(unsigned int i=0; i<query_nb; i++){
    //     printf("%d, ", h_count[i]);
    //     // assert(h_count[i] >= k);
    // }
    // printf("\n");

    // do insertion sort on cpu exploiting h_candidates
    
    for(unsigned int query_index=0; query_index<query_nb; query_index++){

        // do insertion sort
        // masked_insertion_sort(dist, index, h_candidates, ref_nb, k, query_index, query_nb);
        // insertion_sort(dist, index, ref_nb, k);
        selection_sort(h_gpu_dist, h_gpu_index, h_candidates, ref_nb, k, query_index, query_nb, knn_dist, knn_index);
        
        // print first k elements
        // printf("Query %d: ", query_index);
        // for(unsigned int i=0; i<k; i++){
        //     printf("%f || ", dist[i]);
        // }
        // printf("\n");

        // copy k smallest distances and their associated index (done in selection_sort)
        // for (int j=0; j<k; ++j) {
        //     knn_dist[j * query_nb + query_index]  = dist[j];
        //     knn_index[j * query_nb + query_index] = index[j];
        // }
    }

   return true;
}






// solution to test if computing min and counting candidates to help cpu actually helps the cpu
bool your_solution_only_dist(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,    // output fields
                     int *         knn_index) {

    uint64_t o_matrix_size = 1LL * ref_nb * query_nb * sizeof(float);

    float * cpu_dist   = (float*) malloc(o_matrix_size);
    float * h_gpu_dist = (float*) malloc(o_matrix_size);
    int   * h_gpu_index    = (int*)   malloc(o_matrix_size);

    int blockSize = dim;        // Number of threads per block
    int gridSize = ref_nb;      // Number of blocks

    // printf("blockSize: %d\n", blockSize);
    // printf("gridSize: %d\n", gridSize);

    // copy ref and query into cuda mem
    float   *d_ref, *d_query;
    float   *d_gpu_dist;
    int     *d_index;

    hipMalloc(&d_ref, ref_nb * dim * sizeof(float));
    hipMalloc(&d_query, ref_nb * dim * sizeof(float));

    hipMalloc(&d_gpu_dist, o_matrix_size);
    hipMalloc(&d_index, o_matrix_size);

    hipMemcpy(  d_ref,   ref, ref_nb * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_query, query, ref_nb * dim * sizeof(float), hipMemcpyHostToDevice);


    // Calculate the next power of 2 for dim
    int nextPow2 = 1;
    while (nextPow2 < dim) {
        nextPow2 <<= 1;
    }

    // Calculate the number of elements required in smem with padding
    int paddedDim = nextPow2/2;
    int smemSize = 3 * paddedDim * sizeof(float);
    // printf("paddedDim: %d\n", paddedDim);

    for(unsigned int query_index=0; query_index<query_nb; query_index++){
        // printf("Query %d\n", query_index);
        padded_cdist<<< gridSize, paddedDim, smemSize >>>(d_ref, ref_nb, d_query, query_nb, dim, paddedDim, query_index, d_gpu_dist, d_index);
    }

    // mem copy back to cpu
    // very expensive and can be done in parallel with k selection, as long as memcopy is done before actual selection
    hipMemcpy(h_gpu_dist, d_gpu_dist, o_matrix_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_gpu_index, d_index, o_matrix_size, hipMemcpyDeviceToHost);

    hipFree(d_ref);
    hipFree(d_query);


    // K SELECTION ON CPU --------------------------------------------------------------------------------------------------------------------------------

    for(int i=0; i<query_nb; i++){
        insertion_sort_on_matrix(h_gpu_dist, h_gpu_index, ref_nb, k, i, query_nb);

        // Copy k smallest distances and their associated index
        for (int j=0; j<k; ++j) {
        knn_dist[j * query_nb + i]  = h_gpu_dist[(j*query_nb)+i];
        knn_index[j * query_nb + i] = h_gpu_index[(j*query_nb)+i];
        }
    }

   return true;
}